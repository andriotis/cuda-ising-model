#include "hip/hip_runtime.h"
#include <stdio.h>
#include "parallel.h"
#include "utils.h"
#include "math.h"

int *parallel_ising(int *G, int n, int b, int k) {
    int *I;
    allocate_model(&I, n);
    copy_model(G, I, n);

    int *gpu_I, *gpu_P, *gpu_O;
    hipMalloc((void **)&gpu_I, n * n * sizeof(int));
    hipMalloc((void **)&gpu_P, (n + 2) * (n + 2) * sizeof(int));
    hipMalloc((void **)&gpu_O, n * n * sizeof(int));

    hipMemcpy(gpu_I, I, n * n * sizeof(int), hipMemcpyHostToDevice); 

    dim3 block_dim = dim3(1, 1);
    dim3 grid_dim = dim3(ceil(((float)n)/b), ceil(((float)n)/b));

    for (int step = 0; step < k; step++) {
        parallel_pad<<<grid_dim, block_dim>>>(gpu_P, gpu_I, n, b);
        parallel_update<<<grid_dim, block_dim>>>(gpu_P, gpu_O, n, b);
        swap(&gpu_I, &gpu_O);
    }

    hipMemcpy(I, gpu_I, n * n * sizeof(int), hipMemcpyDeviceToHost);

    return I;
}

__global__ void parallel_pad(int *P, int *G, int n, int b) {

    int start_row = blockIdx.y * b;
    int end_row = start_row + b;
    int start_col = blockIdx.x * b;
    int end_col = start_col + b;

    for (int row = start_row; row < end_row; row++) {
        for (int col = start_col; col < end_col; col++) {
            if (row < n && col < n) {
                P[(n + 2) * (row + 1) + (col + 1)] = G[n * row + col];
                if (row == 0)
                    P[(n + 2) * (n + 1) + (col + 1)] = G[n * row + col];
                if (row == n - 1)
                    P[col + 1] = G[n * row + col];
                if (col == 0)
                    P[(n + 2) * (row + 1) + (n + 1)] = G[n * row + col];
                if (col == n - 1)
                    P[(n + 2) * (row + 1)] = G[n * row + col];
            }
        }
    }
}

__global__ void parallel_update(int *P, int *O, int n, int b) {

    int start_row = blockIdx.y * b;
    int end_row = start_row + b;
    int start_col = blockIdx.x * b;
    int end_col = start_col + b;

    for (int row = start_row; row < end_row; row++) {
        for (int col = start_col; col < end_col; col++) {
            if (row < n && col < n) {
                O[n * row + col] = parallel_moment(P, row + 1, col + 1, n + 2);
            }
        }
    }
}

__device__ int parallel_moment(int *P, int i, int j, int n) {
    int moment = P[n * (i - 1) + j]
               + P[n * i + (j - 1)]
               + P[n * i + j]
               + P[n * (i + 1) + j]
               + P[n * i + (j + 1)];

    return (moment > 0) ? 1 : -1;
}
