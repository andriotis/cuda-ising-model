#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "shared.h"
#include "utils.h"
#include "math.h"

int *shared_ising(int *G, int n, int b, int k) {
    int *I;
    allocate_model(&I, n);
    copy_model(G, I, n);
    
    int *gpu_I, *gpu_P, *gpu_O;
    hipMalloc((void **)&gpu_I, n * n * sizeof(int));
    hipMalloc((void **)&gpu_P, (n + 2) * (n + 2) * sizeof(int));
    hipMalloc((void **)&gpu_O, n * n * sizeof(int));

    hipMemcpy(gpu_I, I, n * n * sizeof(int), hipMemcpyHostToDevice); 

    dim3 block_dim = dim3(1, 1);
    dim3 grid_dim = dim3(ceil(((float)n)/b), ceil(((float)n)/b));

    for (int step = 0; step < k; step++) {
        shared_pad<<<grid_dim, block_dim>>>(gpu_P, gpu_I, n, b);
        shared_update<<<grid_dim, block_dim, (b + 2) * (b + 2) * sizeof(int)>>>(gpu_P, gpu_O, n, b);
        swap(&gpu_I, &gpu_O);
    }

    hipMemcpy(I, gpu_I, n * n * sizeof(int), hipMemcpyDeviceToHost);

    return I;
}

__global__ void shared_pad(int *P, int *G, int n, int b) {

    int start_row = blockIdx.y * b;
    int end_row = start_row + b;
    int start_col = blockIdx.x * b;
    int end_col = start_col + b;

    for (int row = start_row; row < end_row; row++) {
        for (int col = start_col; col < end_col; col++) {
            if (row < n && col < n) {
                P[(n + 2) * (row + 1) + (col + 1)] = G[n * row + col];
                if (row == 0)
                    P[(n + 2) * (n + 1) + (col + 1)] = G[n * row + col];
                if (row == n - 1)
                    P[col + 1] = G[n * row + col];
                if (col == 0)
                    P[(n + 2) * (row + 1) + (n + 1)] = G[n * row + col];
                if (col == n - 1)
                    P[(n + 2) * (row + 1)] = G[n * row + col];
            }
        }
    }
}

__global__ void shared_update(int *P, int *O, int n, int b) {

    extern __shared__ int shared[];

    int start_row = blockIdx.y * b;
    int end_row = start_row + b;
    int start_col = blockIdx.x * b;
    int end_col = start_col + b;

    for (int i = 0; i < b + 2; i++) {
        for (int j = 0; j < b + 2; j++) {
            if (start_row + i < n + 2 && start_col + j < n + 2) {
                shared[i * (b + 2) + j] = P[(n + 2) * (start_row + i) + (start_col + j)];
            }
        }
    }

    for (int i = start_row; i < end_row; i++) {
        for (int j = start_col; j < end_col; j++) {
            if (i < n && j < n) {
                O[n * i + j] = shared_moment(shared, i + 1 - start_row, j + 1 - start_col, b + 2);
            }
        }
    }
}

__device__ int shared_moment(int *P, int i, int j, int n) {
    int moment = P[n * (i - 1) + j]
               + P[n * i + (j - 1)]
               + P[n * i + j]
               + P[n * (i + 1) + j]
               + P[n * i + (j + 1)];

    return (moment > 0) ? 1 : -1;
}