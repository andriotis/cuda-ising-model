#include "hip/hip_runtime.h"
#include "v1.h"
#include "utils.h"

int *get_v1(int *G, int n, int k) {

    int *gpu_G, *gpu_P, *gpu_O;
    hipMalloc((void **)&gpu_G, n * n * sizeof(int));
    hipMalloc((void **)&gpu_P, (n + 2) * (n + 2) * sizeof(int));
    hipMalloc((void **)&gpu_O, n * n * sizeof(int));

    hipMemcpy(gpu_G, G, n * n * sizeof(int), hipMemcpyHostToDevice);

    int block_size = 32;
    dim3 block_dim = dim3(block_size, block_size);

    int grid_size = (n + block_size - 1) / block_size;
    dim3 grid_dim = dim3(grid_size, grid_size);

    for (int step = 0; step < k; step++) {
        pad_v1<<<grid_dim, block_dim>>>(gpu_P, gpu_G, n);
        update_v1<<<grid_dim, block_dim>>>(gpu_P, gpu_O, n);
        swap(&gpu_G, &gpu_O);
    }

    hipMemcpy(G, gpu_G, n * n * sizeof(int), hipMemcpyDeviceToHost);

    return G;
}

__global__ void pad_v1(int *P, int *G, int n) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < n && j < n) {
        P[(n + 2) * (i + 1) + (j + 1)] = G[n * i + j];
        if (i == 0)
            P[(n + 2) * (n + 1) + (j + 1)] = G[n * i + j];
        if (i == n - 1)
            P[j + 1] = G[n * i + j];
        if (j == 0)
            P[(n + 2) * (i + 1) + (n + 1)] = G[n * i + j];
        if (j == n - 1)
            P[(n + 2) * (i + 1)] = G[n * i + j];
    }
}

__global__ void update_v1(int *P, int *O, int n) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < n && j < n)
        moment_v1(P, O, i + 1, j + 1, n);
}

__device__ void moment_v1(int *P, int *O, int i, int j, int n) {
    int moment = P[(n + 2) * (i - 1) + j]
               + P[(n + 2) * i + (j - 1)]
               + P[(n + 2) * i + j]
               + P[(n + 2) * (i + 1) + j]
               + P[(n + 2) * i + (j + 1)];

    if (moment > 0)
        O[n * (i - 1) + (j - 1)] = 1;
    else
        O[n * (i - 1) + (j - 1)] = -1;
}