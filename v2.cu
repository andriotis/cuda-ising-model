#include "hip/hip_runtime.h"
#include <stdio.h>

#include "v2.h"
#include "math.h"
#include "utils.h"

int *get_v2(int *G, int n, int b, int k) {
    int *I;
    allocate_model(&I, n);
    copy_model(G, I, n);

    int *gpu_I, *gpu_P, *gpu_O;
    hipMalloc((void **)&gpu_I, n * n * sizeof(int));
    hipMalloc((void **)&gpu_P, (n + 2) * (n + 2) * sizeof(int));
    hipMalloc((void **)&gpu_O, n * n * sizeof(int));

    hipMemcpy(gpu_I, I, n * n * sizeof(int), hipMemcpyHostToDevice); 

    dim3 block_dim = dim3(1, 1);
    dim3 grid_dim = dim3(ceil(((float)n)/b), ceil(((float)n)/b));

    for (int step = 0; step < k; step++) {
        pad_v2<<<grid_dim, block_dim>>>(gpu_P, gpu_I, n, b);
        update_v2<<<grid_dim, block_dim>>>(gpu_P, gpu_O, n, b);
        swap(&gpu_I, &gpu_O);
    }

    hipMemcpy(I, gpu_I, n * n * sizeof(int), hipMemcpyDeviceToHost);

    return I;
}

__global__ void pad_v2(int *P, int *G, int n, int b) {

    int start_row = blockIdx.y * b;
    int end_row = start_row + b;
    int start_col = blockIdx.x * b;
    int end_col = start_col + b;

    for (int row = start_row; row < end_row; row++) {
        for (int col = start_col; col < end_col; col++) {
            if (row < n && col < n) {
                P[(n + 2) * (row + 1) + (col + 1)] = G[n * row + col];
                if (row == 0)
                    P[(n + 2) * (n + 1) + (col + 1)] = G[n * row + col];
                if (row == n - 1)
                    P[col + 1] = G[n * row + col];
                if (col == 0)
                    P[(n + 2) * (row + 1) + (n + 1)] = G[n * row + col];
                if (col == n - 1)
                    P[(n + 2) * (row + 1)] = G[n * row + col];
            }
        }
    }
}

__global__ void update_v2(int *P, int *O, int n, int b) {

    int start_row = blockIdx.y * b;
    int end_row = start_row + b;
    int start_col = blockIdx.x * b;
    int end_col = start_col + b;

    for (int row = start_row; row < end_row; row++) {
        for (int col = start_col; col < end_col; col++) {
            if (row < n && col < n) {
                moment_v2(P, O, row + 1, col + 1, n);
            }
        }
    }
}

__device__ void moment_v2(int *P, int *O, int i, int j, int n) {

    int moment = P[(n + 2) * (i - 1) + j]
               + P[(n + 2) * i + (j - 1)]
               + P[(n + 2) * i + j]
               + P[(n + 2) * (i + 1) + j]
               + P[(n + 2) * i + (j + 1)];

    if (moment > 0)
        O[n * (i - 1) + (j - 1)] = 1;
    else
        O[n * (i - 1) + (j - 1)] = -1;
}